#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include "img_precess_k.hpp"
#include <iostream>

// src_crop--crop+rescale-->dst_crop, hwc--permute-->chw, bgr--shuffle-->rgb, uint8 or float--normalize--->float
template <typename T>
static void __global__ crop_scale_permute_shuffle_normalize(const T* src, float* channel_r, float* channel_g, float* channel_b, const PreParam param) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i > param.dst_crop.w || j > param.dst_crop.h) {return;}
    unsigned dst_ind = j*param.dst_step+i;
    float3 rgb;
    
    float scaled_i = param.scale_inv.x * i + param.src_crop.l;
    float scaled_j = param.scale_inv.y * j + param.src_crop.t;
    float f_scaled_i = floorf(scaled_i);
    float f_scaled_j = floorf(scaled_j);
    unsigned src_i = scaled_i - f_scaled_i < 0.5 ? f_scaled_i : f_scaled_i+1;
    unsigned src_j = scaled_j - f_scaled_j < 0.5 ? f_scaled_j : f_scaled_j+1;
    src_i = src_i >= param.src_size.x ? param.src_size.x-1 : src_i;
    src_j = src_j >= param.src_size.y ? param.src_size.y-1 : src_j;
    unsigned src_ind = (src_j*param.src_step+src_i)*3;
    rgb.x = (src[src_ind+2]*1.0f - param.mean.x)*param.norm_inv.x;
    rgb.y = (src[src_ind+1]*1.0f - param.mean.y)*param.norm_inv.y;
    rgb.z = (src[src_ind+0]*1.0f - param.mean.z)*param.norm_inv.z;
    channel_r[dst_ind] = rgb.x;
    channel_g[dst_ind] = rgb.y;
    channel_b[dst_ind] = rgb.z;
}

// src_crop--crop+rescale-->dst_crop, hwc--permute-->chw, bgr--shuffle-->rgb, half--normalize--->half
static void __global__ crop_scale_permute_shuffle_normalize(const half* src, half* channel_r, half* channel_g, half* channel_b, const PreParam param) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i > param.dst_crop.w || j > param.dst_crop.h) {return;}
    unsigned dst_ind = j*param.dst_step+i;
    
    float scaled_i = param.scale_inv.x * i + param.src_crop.l;
    float scaled_j = param.scale_inv.y * j + param.src_crop.t;
    float f_scaled_i = floorf(scaled_i);
    float f_scaled_j = floorf(scaled_j);
    unsigned src_i = scaled_i - f_scaled_i < 0.5 ? f_scaled_i : f_scaled_i+1;
    unsigned src_j = scaled_j - f_scaled_j < 0.5 ? f_scaled_j : f_scaled_j+1;
    src_i = src_i >= param.src_size.x ? param.src_size.x-1 : src_i;
    src_j = src_j >= param.src_size.y ? param.src_size.y-1 : src_j;
    unsigned src_ind = (src_j*param.src_step+src_i)*3;
    channel_r[dst_ind] = __hmul(__hsub(src[src_ind+2], param.mean16.x), param.norm_inv16.x);
    channel_g[dst_ind] = __hmul(__hsub(src[src_ind+1], param.mean16.y), param.norm_inv16.y);
    channel_b[dst_ind] = __hmul(__hsub(src[src_ind+0], param.mean16.z), param.norm_inv16.z);
}

template <typename T0, typename T1>
hipError_t launch_pre_kernel(const T0* d_mat_src, T1* d_mat_dst, const PreParam& param, void* stream_ptr) {
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);
    dim3 threads(16,16);
    dim3 blocks((param.dst_crop.w+threads.x-1)/threads.x, (param.dst_crop.h+threads.y-1)/threads.y);    // a thread for a output pixel
    unsigned int planar_size = param.dst_size.x*param.dst_size.y;
    unsigned int lt_loc = param.dst_crop.t * param.dst_step + param.dst_crop.l;
    crop_scale_permute_shuffle_normalize<<<blocks,threads,0,stream>>>(d_mat_src, d_mat_dst+lt_loc, d_mat_dst+planar_size+lt_loc, d_mat_dst+planar_size*2+lt_loc, param);
    hipError_t err = hipGetLastError();
    return err;
}
template hipError_t launch_pre_kernel<uint8_t, float>(const uint8_t* d_mat_src, float* d_mat_dst, const PreParam& param, void* stream_ptr);
template hipError_t launch_pre_kernel<float, float>(const float* d_mat_src, float* d_mat_dst, const PreParam& param, void* stream_ptr);
template hipError_t launch_pre_kernel<half, half>(const half* d_mat_src, half* d_mat_dst, const PreParam& param, void* stream_ptr);

// multi_input--pre_kernel-->multi_output
template <typename T0, typename T1>
hipError_t launch_batched_pre_kernel(const std::vector<T0*>& d_mat_src_vec, T1* d_mat_dst, const PreParam& param, void* stream_ptr) {
    unsigned int dst_mat_size = param.dst_size.y*param.dst_step*3*sizeof(T1);
    for (const auto& d_mat_src: d_mat_src_vec) {
        launch_pre_kernel(d_mat_src, d_mat_dst, param, stream_ptr);
        d_mat_dst += dst_mat_size;
    }
    hipError_t err = hipGetLastError();
    return err;
}
template hipError_t launch_batched_pre_kernel<uint8_t, float>(const std::vector<uint8_t*>& d_mat_src_vec, float* d_mat_dst, const PreParam& param, void* stream_ptr);
template hipError_t launch_batched_pre_kernel<float, float>(const std::vector<float*>& d_mat_src_vec, float* d_mat_dst, const PreParam& param, void* stream_ptr);

// multi_input--pre_kernel-->multi_output
template <typename T0, typename T1>
hipError_t launch_batched_pre_kernel(const T0* d_mat_src, T1* d_mat_dst, const PreParam& param, void* stream_ptr) {
    // unsigned int src_mat_size = param.src_size.y*param.src_step*3*sizeof(T0);
    // unsigned int dst_mat_size = param.dst_size.y*param.dst_step*3*sizeof(T1); // cause error, just a reminder
    unsigned int src_mat_size = param.src_size.y*param.src_step*3;
    unsigned int dst_mat_size = param.dst_size.y*param.dst_step*3;
    for (unsigned i = 0; i < param.batch_size; i++) {
        launch_pre_kernel(d_mat_src, d_mat_dst, param, stream_ptr);
        d_mat_src += src_mat_size;
        d_mat_dst += dst_mat_size;
    }
    hipError_t err = hipGetLastError();
    return err;
}
template hipError_t launch_batched_pre_kernel<uint8_t, float>(const uint8_t* d_mat_src, float* d_mat_dst, const PreParam& param, void* stream_ptr);
template hipError_t launch_batched_pre_kernel<float, float>(const float* d_mat_src, float* d_mat_dst, const PreParam& param, void* stream_ptr);
template hipError_t launch_batched_pre_kernel<half, half>(const half* d_mat_src, half* d_mat_dst, const PreParam& param, void* stream_ptr);
